#include "hip/hip_runtime.h"
texture<float, 1, hipReadModeElementType> texRef;

__global__ void actiune_thread(float* a_d, float* b_d,float *r_d,int N);

// Kernelul ce se executa pe device-ul CUDA 
__global__ void actiune_thread(float* a_d, float* b_d,float *r_d,int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	//if(idx < N){
	
		r_d[idx] = - tex1Dfetch(texRef, idx);
	
	//}
	
	
}

extern "C"
hipError_t launch_actiune_thread(float* a_d, float* b_d,float *r_d,int N,dim3 DIM_GRID, dim3 DIM_BLOCK)
{

	actiune_thread <<<DIM_GRID, DIM_BLOCK>>> (a_d, b_d,r_d,N);
	
	return hipGetLastError();
}

extern "C"
hipError_t legare(size_t * offset,const void * devPtr,size_t size = UINT_MAX){

	hipBindTexture(offset, texRef, devPtr, UINT_MAX); 
	
	return hipGetLastError();

}

extern "C"
hipError_t release(){

	hipUnbindTexture(texRef);

	return hipGetLastError();

}